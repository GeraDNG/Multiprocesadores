#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------------

*

* Multiprocesadores: Proyecto final.

* Fecha: 01 de diciembre del 2019.

* Autor: Gerardo Daniel Naranjo Gallegos, A01209499.

* Implementación: CUDA.

*

*-----------------------------------------------------------------------------*/


// ---------------------------------------------------------------------------//
//																LIBRERÍAS																		//
// ---------------------------------------------------------------------------//

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "utils/cheader.h"

// ---------------------------------------------------------------------------//
//										DEFINIR VARIABLES QUE NO CAMBIAN												//
// ---------------------------------------------------------------------------//

#define MIN(vector1,vector2) (vector1<vector2?vector1:vector2)
#define SIZE	100000000
#define THREADS	256
#define BLOCKS	MIN(32, (SIZE + THREADS - 1)/ THREADS)

// ---------------------------------------------------------------------------//
//																FUNCIÓN																		  //
// ---------------------------------------------------------------------------//

__global__ void calculate(int *resta, int *suma, int *vector1, int *vector2) {
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	if (i < SIZE) {
		suma[i] = vector1[i] + vector2[i];
		resta[i] = vector1[i] - vector2[i];
	}
}

// ---------------------------------------------------------------------------//
//																	MAIN																			//
// ---------------------------------------------------------------------------//


int main(int argc, char const *argv[]) {
	// Definir variables a utilizar:
	int *vector1, *vector2, *suma, *resta;
	int *device_vector1, *device_vector2, *device_suma, *device_resta;
	double ms = 0.0;

	// Inicializar variables:
	vector1 = (int *) malloc(sizeof(int) * SIZE);
	vector2 = (int *) malloc(sizeof(int) * SIZE);
	suma = (int *) malloc(sizeof(int) * SIZE);
	resta = (int *) malloc(sizeof(int) * SIZE);

	// Rellenar vectores utilizados:
	random_array(vector1, SIZE);
	fill_array(vector2, SIZE);

	// Mostrar ambos vectores:
	display_array("a", vector1);
	display_array("b", vector2);

	// Asignar memoria a las variables en el dispositivo (GPU):
	hipMalloc((void**) &device_vector1, SIZE * sizeof(int));
	hipMalloc((void**) &device_vector2, SIZE * sizeof(int));
	hipMalloc((void**) &device_suma, SIZE * sizeof(int));
	hipMalloc((void**) &device_resta, SIZE * sizeof(int));

	// Copiar vectores a vectores en el dispositivo (GPU):
	hipMemcpy(device_vector1, vector1, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_vector2, vector2, SIZE * sizeof(int), hipMemcpyHostToDevice);

	// Imprimir mensaje de inicio:
	printf("Iniciando...\n");

	// Método FOR para ejecutar la parte en paralelo en la GPU:
	for (int j = 0; j < N; j++) {
		start_timer();
		calculate<<<BLOCKS, THREADS>>>(device_resta, device_suma, device_vector1, device_vector2);
		ms += stop_timer();
	}

	// Copiar vectores resultantes del GPU al CPU:
	hipMemcpy(suma, device_suma, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(resta, device_resta, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	// Imprimir/Mostrar en la terminal los vectores resultantes:
	display_array("c", suma);
	display_array("d", resta);

	// Imprimir/Mostrar en terminal el tiempo de ejecución del programa:
	printf("Tiempo de ejecución = %.5lf ms\n", (ms / N));

	// Liberar la memoria utilizada en el GPU por las variables:
	hipFree(device_vector1);
	hipFree(device_vector2);
	hipFree(device_suma);
	hipFree(device_resta);

	// Liberar la memoria del CPU utilizada por las variables:
	free(vector1);
	free(vector2);
	free(suma);
	free(resta);

	// Fin del programa:
	return 0;
}
