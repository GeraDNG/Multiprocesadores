#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int *c, int *a, int *b) {
	*c = *a + *b;
}

int main(int argc, char* argv[]) {
	int a, b, c;
	int *da, *db, *dc;
	
	hipMalloc((void**) &da, sizeof(int));
	hipMalloc((void**) &db, sizeof(int));
	hipMalloc((void**) &dc, sizeof(int));
	
	scanf("%i %i", &a, &b);
	
	hipMemcpy(da, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(db, &b, sizeof(int), hipMemcpyHostToDevice);
	
	add<<<1, 1>>>(dc, da, db);
	
	hipMemcpy(&c, dc, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("c = %i\n", c);
	
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	
	return 0;
}
